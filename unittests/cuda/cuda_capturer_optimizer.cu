#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cudaflow.hpp>

#include "./simple_graph/graph_executor.hpp"
#include "./simple_graph/tree.hpp"
#include "./simple_graph/random_DAG.hpp"
#include "./simple_graph/tree.hpp"
#include "./simple_graph/diamond.hpp"

// ----------------------------------------------------------------------------
// Graph traversal
// ----------------------------------------------------------------------------
template <typename GRAPH, typename OPT, typename... OPT_Args>
void traversal(OPT_Args&&... args) {
  for(int i = 0; i < 13; ++i) {
    Graph* g;
    if constexpr(std::is_same_v<GRAPH, Tree>) {
      g = new Tree(::rand() % 3 + 1, ::rand() % 4 + 1);
    }
    else if constexpr(std::is_same_v<GRAPH, RandomDAG>) {
      g = new RandomDAG(::rand() % 7 + 1, ::rand() % 4 + 1, ::rand() % 3 + 1);
    }
    else if constexpr(std::is_same_v<GRAPH, Diamond>) {
      g = new Diamond(::rand() % 5 + 1, ::rand() % 4 + 1);
    }
    GraphExecutor<OPT> executor(*g, 0); 
    executor.traversal(std::forward<OPT_Args>(args)...);

    REQUIRE(g->traversed());
    delete g;
  }

}

TEST_CASE("cudaCapturer.tree.Sequential") {
  traversal<Tree,tf::SequentialOptimizer>();
}

TEST_CASE("cudaCapturer.tree.RoundRobin") {
  traversal<Tree, tf::RoundRobinOptimizer>(4);
}

TEST_CASE("cudaCapturer.randomDAG.Sequential") {
  traversal<RandomDAG,tf::SequentialOptimizer>();
}

TEST_CASE("cudaCapturer.randomDAG.RoundRobin") {
  traversal<RandomDAG, tf::RoundRobinOptimizer>(4);
}

TEST_CASE("cudaCapturer.diamond.Sequential") {
  traversal<Diamond,tf::SequentialOptimizer>();
}

TEST_CASE("cudaCapturer.diamond.RoundRobin") {
  traversal<Diamond, tf::RoundRobinOptimizer>(4);
}

//------------------------------------------------------
//dependencies
//------------------------------------------------------

template <typename OPT, typename... OPT_Args>
void dependencies(OPT_Args ...args) {
  
  for(int t = 0; t < 17; ++t) {
    int num_partitions = ::rand() % 5 + 1;
    int num_iterations = ::rand() % 7 + 1;

    Diamond g(num_partitions, num_iterations);

    tf::cudaFlowCapturer cf;
    cf.make_optimizer<OPT>(std::forward<OPT_Args>(args)...);

    int* inputs{nullptr};
    REQUIRE(hipMallocManaged(&inputs, num_partitions * sizeof(int)) == hipSuccess);
    REQUIRE(hipMemset(inputs, 0, num_partitions * sizeof(int)) == hipSuccess);

    std::vector<std::vector<tf::cudaTask>> tasks;
    tasks.resize(g.get_size());

    for(size_t l = 0; l < g.get_size(); ++l) {
      tasks[l].resize((g.get_graph())[l].size());
      for(size_t i = 0; i < (g.get_graph())[l].size(); ++i) {
        
        //if(l % 2 == 1) {
          //tasks[l][i] = cf.single_task([inputs, i] __device__ () {
            //inputs[i]++;
          //});
        //}
        //else {
          tasks[l][i] = cf.for_each(
            inputs, inputs + num_partitions, 
            [] __device__(int& v) { v *= 2; }
          );
        //}
      }
    }

    for(size_t l = 0; l < g.get_size() - 1; ++l) {
      for(size_t i = 0; i < (g.get_graph())[l].size(); ++i) {
        for(auto&& out_node: g.at(l, i).out_nodes) {
          tasks[l][i].precede(tasks[l + 1][out_node]);
        }
      }
    }

    cf.offload();
    
    int result = 2;
    for(int i = 1; i < num_iterations; ++i) {
      result = result * 2 + 2;
    }

  //for(int i = 0; i < num_partitions; ++i) {
      //REQUIRE(inputs[i] == result);
    //}

    REQUIRE(hipFree(inputs) == hipSuccess);
  }
}

//TEST_CASE("cudaCapturer.dependencies.diamond.Sequential") {
  //dependencies<tf::SequentialOptimizer>();
//}

//TEST_CASE("cudaCapturer.dependencies.diamond.RoundRobin") {
  //dependencies<tf::RoundRobinOptimizer>(4);
//}
